#include "hip/hip_runtime.h"
#include "OrderParameterMeshGPU.cuh"

//! Implements workaround atomic float addition on sm_1x hardware
__device__ inline void atomicFloatAdd(float* address, float value)
    {
#if (__CUDA_ARCH__ < 200)
    float old = value;
    float new_old;
    do
        {
        new_old = atomicExch(address, 0.0f);
        new_old += old;
        }
    while ((old = atomicExch(address, new_old))!=0.0f);
#else
    atomicAdd(address, value);
#endif
    }


/*! \param x Distance on mesh in units of the mesh size
 */
__device__ inline Scalar assignTSC(Scalar x)
    {
    Scalar xsq = x*x;
    Scalar fac =(Scalar(3.0/2.0)-copysignf(x,Scalar(1.0)));

    Scalar ret(0.0);
    if (xsq <= Scalar(1.0/4.0))
        ret = Scalar(3.0/4.0) - xsq;
    else if (xsq <= Scalar(9.0/4.0))
        ret = Scalar(1.0/2.0)*fac*fac;

    return ret;
    }

/*! \param x Distance on mesh in units of the mesh size
 */
__device__ inline Scalar assignTSCderiv(Scalar x)
    {
    Scalar xsq = x*x;
    Scalar xabs = copysignf(x,Scalar(1.0));
    Scalar fac =(Scalar(3.0/2.0)-xabs);

    Scalar ret(0.0);
    if (xsq <= Scalar(1.0/4.0))
        ret = -Scalar(2.0)*x;
    else if (xsq <= Scalar(9.0/4.0))
        ret = -fac*x/xabs;

    return ret;
    }


__device__ int3 find_cell(const Scalar3& pos,
                           const unsigned int& inner_nx,
                           const unsigned int& inner_ny,
                           const unsigned int& inner_nz,
                           const uint3& n_ghost_cells,
                           const BoxDim& box
                           )
    {
    // compute coordinates in units of the mesh size
    Scalar3 f = box.makeFraction(pos);
    uchar3 periodic = box.getPeriodic();

    Scalar3 reduced_pos = make_scalar3(f.x * (Scalar)inner_nx,
                                       f.y * (Scalar)inner_ny,
                                       f.z * (Scalar)inner_nz);

    reduced_pos += make_scalar3(n_ghost_cells.x, n_ghost_cells.y, n_ghost_cells.z);

    // find cell the particle is in (rounding up/down)
    int ix = ((reduced_pos.x >= 0) ? reduced_pos.x : (reduced_pos.x - Scalar(1.0)));
    int iy = ((reduced_pos.y >= 0) ? reduced_pos.y : (reduced_pos.y - Scalar(1.0)));
    int iz = ((reduced_pos.z >= 0) ? reduced_pos.z : (reduced_pos.z - Scalar(1.0)));

    // handle particles on the boundary
    if (periodic.x && ix == (int)inner_nx)
        ix = 0;
    if (periodic.y && iy == (int)inner_ny)
        iy = 0;
    if (periodic.z && iz == (int)inner_nz)
        iz = 0;

    return make_int3(ix, iy, iz);
    }

__global__ void gpu_bin_particles_kernel(const unsigned int N,
                                         const Scalar4 *d_postype,
                                         Scalar4 *d_particle_bins,
                                         unsigned int *d_n_cell,
                                         unsigned int *d_overflow,
                                         const Index2D bin_idx,
                                         const uint3 mesh_dim,
                                         const uint3 n_ghost_bins,
                                         const Scalar *d_mode,
                                         const BoxDim box)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    Scalar4 postype = d_postype[idx];

    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int type = __float_as_int(postype.w);
    Scalar mode = d_mode[type];

    int3 bin_dim = make_int3(mesh_dim.x+2*n_ghost_bins.x,
                             mesh_dim.y+2*n_ghost_bins.y,
                             mesh_dim.z+2*n_ghost_bins.z);

    // compute coordinates in units of the cell size
    int3 bin_coord = find_cell(pos, mesh_dim.x, mesh_dim.y, mesh_dim.z, n_ghost_bins, box);

    // ignore particles that are not within our domain (the error should be caught by HOOMD's cell list)
    if (bin_coord.x < 0 || bin_coord.x >= bin_dim.x ||
        bin_coord.y < 0 || bin_coord.y >= bin_dim.y ||
        bin_coord.z < 0 || bin_coord.z >= bin_dim.z) return;

    // row-major mapping of bins onto array
    unsigned int bin = bin_coord.x + bin_dim.x * (bin_coord.y + bin_dim.y * bin_coord.z);

    // we bin non-deterministically here
    unsigned int n = atomicInc(&d_n_cell[bin], 0xffffffff);

    if (n >= bin_idx.getH())
        {
        // overflow
        atomicMax(d_overflow, n+1);
        return;
        }

    // store distance to bin center in bin in units of bin size
    Scalar3 f = box.makeFraction(pos);
    f = f*make_scalar3(mesh_dim.x, mesh_dim.y, mesh_dim.z);
    Scalar3 c = make_scalar3((Scalar)(bin_coord.x - (int)n_ghost_bins.x) + Scalar(0.5),
                             (Scalar)(bin_coord.y - (int)n_ghost_bins.y) + Scalar(0.5),
                             (Scalar)(bin_coord.z - (int)n_ghost_bins.z) + Scalar(0.5));
    Scalar3 shift = f - c;

    d_particle_bins[bin_idx(bin,n)] = make_scalar4(shift.x,shift.y,shift.z, mode);
    }

void gpu_bin_particles(const unsigned int N,
                       const Scalar4 *d_postype,
                       Scalar4 *d_particle_bins,
                       unsigned int *d_n_cell,
                       unsigned int *d_overflow,
                       const Index2D& bin_idx,
                       const uint3 mesh_dim,
                       const uint3 n_ghost_bins,
                       const Scalar *d_mode,
                       const BoxDim& box)
    {
    unsigned int block_size = 512;

    gpu_bin_particles_kernel<<<N/block_size+1, block_size>>>(N,
             d_postype,
             d_particle_bins,
             d_n_cell,
             d_overflow,
             bin_idx,
             mesh_dim,
             n_ghost_bins,
             d_mode,
             box);
    }

__global__ void gpu_assign_binned_particles_to_scratch_kernel(const uint3 mesh_dim,
                                                           const uint3 n_ghost_bins,
                                                           const Scalar4 *d_particle_bins,
                                                           const unsigned int *d_n_cell,
                                                           Scalar *d_mesh_scratch,
                                                           const Index2D bin_idx,
                                                           const Index2D scratch_idx)
    {
    extern __shared__ Scalar scratch_neighbors[];

    unsigned int bin = blockIdx.x * blockDim.x + threadIdx.x;

    if (bin >= bin_idx.getW()) return;

    int3 bin_dim = make_int3(mesh_dim.x+2*n_ghost_bins.x,
                             mesh_dim.y+2*n_ghost_bins.y,
                             mesh_dim.z+2*n_ghost_bins.z);

    // grid coordinates of bin (row-major)
    int i,j,k;
    k = bin /bin_dim.y / bin_dim.x;
    j = (bin - k * bin_dim.y*bin_dim.x)/bin_dim.x;
    i = bin % bin_dim.x;

    // reset shared memory
    for (unsigned int sidx = 0; sidx < scratch_idx.getH(); ++sidx)
        scratch_neighbors[scratch_idx.getH()*threadIdx.x+sidx] = Scalar(0.0);

    // loop over particles in bin
    unsigned int n_bin = d_n_cell[bin];

    Scalar assign_x, assign_y, assign_z;
    Scalar shift, shift_sq, fac;
    int neigh_bin_idx;
    for (unsigned int idx = 0; idx < n_bin; ++idx)
        {
        Scalar4 xyzm = d_particle_bins[bin_idx(bin,idx)];

        neigh_bin_idx = 0;

        // loop over neighboring bins
        for (int l = -1; l <= 1 ; ++l)
            {
            // precalculate assignment factor
            shift = xyzm.x - (Scalar)l;
            shift_sq = shift*shift;
            fac = (Scalar(3.0/2.0)-copysignf(shift,Scalar(1.0)));

            if (!l)
                assign_x = Scalar(3.0/4.0)-shift_sq;
            else
                assign_x = Scalar(1.0/2.0)*fac*fac;

            for (int m = -1; m <= 1; ++m)
                {
                shift = xyzm.y - (Scalar)m;
                shift_sq = shift*shift;
                fac = (Scalar(3.0/2.0)-copysignf(shift,Scalar(1.0)));

                if (!m)
                    assign_y = Scalar(3.0/4.0)-shift_sq;
                else
                    assign_y = Scalar(1.0/2.0)*fac*fac;

                for (int n = -1; n <= 1; ++n)
                    {
                    shift = xyzm.z - (Scalar)n;
                    shift_sq = shift*shift;
                    fac = (Scalar(3.0/2.0)-copysignf(shift,Scalar(1.0)));

                    if (!n)
                        assign_z = Scalar(3.0/4.0)-shift_sq;
                    else
                        assign_z = Scalar(1.0/2.0)*fac*fac;

                    // compute fraction of particle density assigned to cell
                    // from particles in this bin
                    Scalar mode = xyzm.w;
                    scratch_neighbors[scratch_idx.getH()*threadIdx.x+neigh_bin_idx] += mode*assign_x*assign_y*assign_z;
                    neigh_bin_idx++;
                    }
                }
            } // end of loop over neighboring bins
        } // end of ptl loop

    // write out shared memory to neighboring cells
    // loop over neighboring bins
    neigh_bin_idx = 0;
    bool ignore_x = false;
    bool ignore_y = false;
    bool ignore_z = false;
    for (int l = -1; l <= 1 ; ++l)
        {
        int neighi = i + l;
        if (neighi >= (int)bin_dim.x)
            {
            if (! n_ghost_bins.x)
                neighi = 0;
            else
                ignore_x = true;
            }
        else if (neighi < 0)
            {
            if (! n_ghost_bins.x)
                neighi += (int)bin_dim.x;
            else
                ignore_x = true;
            }

        for (int m = -1; m <= 1; ++m)
            {
            int neighj = j + m;
            if (neighj >= (int) bin_dim.y)
                {
                if (! n_ghost_bins.y)
                    neighj = 0;
                else
                    ignore_y = true;
                }
            else if (neighj < 0)
                {
                if (! n_ghost_bins.y)
                    neighj += (int)bin_dim.y;
                else
                    ignore_y = true;
                }

            for (int n = -1; n <= 1; ++n)
                {
                int neighk = k + n;

                if (neighk >= (int)bin_dim.z)
                    {
                    if (! n_ghost_bins.z)
                        neighk = 0;
                    else
                        ignore_z = true;
                    }
                else if (neighk < 0)
                    {
                    if (! n_ghost_bins.z)
                        neighk += (int)bin_dim.z;
                    else
                        ignore_z = true;
                    }

                if (!ignore_x && !ignore_y && !ignore_z)
                    {
                    uint3 scratch_cell_coord = make_uint3(neighi, neighj, neighk);

                    // write out to global memory
                    unsigned int cell_idx;

                    // use row-major layout
                    cell_idx = scratch_cell_coord.x + bin_dim.x * (scratch_cell_coord.y + bin_dim.y * scratch_cell_coord.z);

                    d_mesh_scratch[scratch_idx(cell_idx,neigh_bin_idx)] =
                        scratch_neighbors[scratch_idx.getH()*threadIdx.x+neigh_bin_idx];
                    }

                ignore_z = false;
                neigh_bin_idx++;
                }
            ignore_y = false;
            }
        ignore_x = false;
        }
    }

__global__ void gpu_reduce_scratch_kernel(const uint3 grid_dim,
                               const Scalar *d_mesh_scratch,
                               const Index2D scratch_idx,
                               hipfftComplex *d_mesh)
    {
    unsigned int cell_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (cell_idx >= grid_dim.x*grid_dim.y*grid_dim.z) return;

    // simply add up contents of scratch cell
    Scalar grid_val(0.0);
    for (unsigned int sidx = 0; sidx < scratch_idx.getH(); ++sidx)
        grid_val += d_mesh_scratch[scratch_idx(cell_idx,sidx)];

    d_mesh[cell_idx].x = grid_val;
    d_mesh[cell_idx].y = Scalar(0.0);
    }

void gpu_assign_binned_particles_to_mesh(const uint3 mesh_dim,
                                         const uint3 n_ghost_bins,
                                         const uint3 grid_dim,
                                         const Scalar4 *d_particle_bins,
                                         Scalar *d_mesh_scratch,
                                         const Index2D& bin_idx,
                                         const Index2D& scratch_idx,
                                         const unsigned int *d_n_cell,
                                         hipfftComplex *d_mesh)
    {
    unsigned int block_size = 128;
    unsigned int n_blocks = bin_idx.getW()/block_size;
    if (bin_idx.getW()%block_size) n_blocks +=1;

    unsigned int shared_size = block_size*scratch_idx.getH()*sizeof(Scalar);

    gpu_assign_binned_particles_to_scratch_kernel<<<n_blocks,block_size,shared_size>>>(
          mesh_dim,
          n_ghost_bins,
          d_particle_bins,
          d_n_cell,
          d_mesh_scratch,
          bin_idx,
          scratch_idx);

    block_size = 512;
    n_blocks = grid_dim.x*grid_dim.y*grid_dim.z/block_size;
    if ((grid_dim.x*grid_dim.y*grid_dim.z)%block_size) n_blocks +=1;
    gpu_reduce_scratch_kernel<<<n_blocks, block_size>>>(grid_dim,
                                                        d_mesh_scratch,
                                                        scratch_idx,
                                                        d_mesh);
    }

__global__ void gpu_compute_mesh_virial_kernel(const unsigned int n_wave_vectors,
                                         hipfftComplex *d_fourier_mesh,
                                         hipfftComplex *d_fourier_mesh_G,
                                         Scalar *d_virial_mesh,
                                         const Scalar3 *d_k,
                                         const Scalar qstarsq,
                                         const bool exclude_dc)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= n_wave_vectors) return;

    if (!exclude_dc || idx != 0)
        {
        // non-zero wave vector
        hipfftComplex f_g = d_fourier_mesh_G[idx];
        hipfftComplex f = d_fourier_mesh[idx];

        Scalar rhog = f_g.x * f.x + f_g.y * f.y;
        Scalar3 k = d_k[idx];

        Scalar ksq = dot(k,k);
        Scalar knorm = sqrtf(ksq);
        Scalar k_cut = sqrtf(qstarsq);
        Scalar fac = expf(-Scalar(12.0)*(knorm/k_cut-Scalar(1.0)));
        Scalar kfac = -Scalar(6.0)/(Scalar(1.0)+fac)/knorm/k_cut;
//        Scalar kfac = -Scalar(1.0/2.0)/qstarsq;
        d_virial_mesh[0*n_wave_vectors+idx] = rhog*kfac*k.x*k.x; // xx
        d_virial_mesh[1*n_wave_vectors+idx] = rhog*kfac*k.x*k.y; // xy
        d_virial_mesh[2*n_wave_vectors+idx] = rhog*kfac*k.x*k.z; // xz
        d_virial_mesh[3*n_wave_vectors+idx] = rhog*kfac*k.y*k.y; // yy
        d_virial_mesh[4*n_wave_vectors+idx] = rhog*kfac*k.y*k.z; // yz
        d_virial_mesh[5*n_wave_vectors+idx] = rhog*kfac*k.z*k.z; // zz
        }
    else
        {
        d_virial_mesh[0*n_wave_vectors+idx] = Scalar(0.0);
        d_virial_mesh[1*n_wave_vectors+idx] = Scalar(0.0);
        d_virial_mesh[2*n_wave_vectors+idx] = Scalar(0.0);
        d_virial_mesh[3*n_wave_vectors+idx] = Scalar(0.0);
        d_virial_mesh[4*n_wave_vectors+idx] = Scalar(0.0);
        d_virial_mesh[5*n_wave_vectors+idx] = Scalar(0.0);
        }
    }

void gpu_compute_mesh_virial(const unsigned int n_wave_vectors,
                             hipfftComplex *d_fourier_mesh,
                             hipfftComplex *d_fourier_mesh_G,
                             Scalar *d_virial_mesh,
                             const Scalar3 *d_k,
                             const Scalar qstarsq,
                             const bool exclude_dc)
    {
    const unsigned int block_size = 512;

    gpu_compute_mesh_virial_kernel<<<n_wave_vectors/block_size+1, block_size>>>(n_wave_vectors,
                                                                          d_fourier_mesh,
                                                                          d_fourier_mesh_G,
                                                                          d_virial_mesh,
                                                                          d_k,
                                                                          qstarsq,
                                                                          exclude_dc);
    }

__global__ void gpu_update_meshes_kernel(const unsigned int n_wave_vectors,
                                         hipfftComplex *d_fourier_mesh,
                                         hipfftComplex *d_fourier_mesh_G,
                                         const Scalar *d_inf_f,
                                         const Scalar3 *d_k,
                                         const unsigned int N_global)
    {
    unsigned int k = blockDim.x * blockIdx.x + threadIdx.x;

    if (k >= n_wave_vectors) return;

    hipfftComplex f = d_fourier_mesh[k];

    // Normalization
    f.x /= (Scalar)N_global;
    f.y /= (Scalar)N_global;
    Scalar val = f.x*f.x+f.y*f.y;

    hipfftComplex fourier_G;
    fourier_G.x =f.x * val * d_inf_f[k];
    fourier_G.y =f.y * val * d_inf_f[k];

    d_fourier_mesh[k] = f;
    d_fourier_mesh_G[k] = fourier_G;
    }

void gpu_update_meshes(const unsigned int n_wave_vectors,
                         hipfftComplex *d_fourier_mesh,
                         hipfftComplex *d_fourier_mesh_G,
                         const Scalar *d_inf_f,
                         const Scalar3 *d_k,
                         const unsigned int N_global)

    {
    const unsigned int block_size = 512;

    gpu_update_meshes_kernel<<<n_wave_vectors/block_size+1, block_size>>>(n_wave_vectors,
                                                                          d_fourier_mesh,
                                                                          d_fourier_mesh_G,
                                                                          d_inf_f,
                                                                          d_k,
                                                                          N_global);
    }

//! Texture for reading particle positions
texture<hipfftComplex, 1, hipReadModeElementType> inv_fourier_mesh_tex;

__global__ void gpu_compute_forces_kernel(const unsigned int N,
                                          const Scalar4 *d_postype,
                                          Scalar4 *d_force,
                                          const Scalar bias,
                                          const uint3 grid_dim,
                                          const uint3 n_ghost_cells,
                                          const Scalar *d_mode,
                                          const BoxDim box,
                                          const Scalar V,
                                          const unsigned int n_global,
                                          const Scalar3 b1,
                                          const Scalar3 b2,
                                          const Scalar3 b3)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    int3 inner_dim = make_int3(grid_dim.x-2*n_ghost_cells.x,
                               grid_dim.y-2*n_ghost_cells.y,
                               grid_dim.z-2*n_ghost_cells.z);

    Scalar4 postype = d_postype[idx];

    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int type = __float_as_int(postype.w);
    Scalar mode = d_mode[type];

    // find cell the particle is in
    int3 cell_coord = find_cell(pos, inner_dim.x, inner_dim.y, inner_dim.z, n_ghost_cells, box);

    // center of cell (in units of the cell size)
    Scalar3 c = make_scalar3((Scalar)(cell_coord.x-(int)n_ghost_cells.x)+Scalar(0.5),
                             (Scalar)(cell_coord.y-(int)n_ghost_cells.y)+Scalar(0.5),
                             (Scalar)(cell_coord.z-(int)n_ghost_cells.z)+Scalar(0.5));

    Scalar3 p = box.makeFraction(pos)*make_scalar3(inner_dim.x, inner_dim.y, inner_dim.z);
    Scalar3 shift_c = p-c;

    Scalar3 force = make_scalar3(0.0,0.0,0.0);

    // assign particle to cell and next neighbors
    Scalar assign_x, assign_y, assign_z;
    Scalar assign_x_deriv, assign_y_deriv, assign_z_deriv;
    Scalar shift, shift_abs,shift_sq, fac;

    for (int i = -1; i <= 1 ; ++i)
        {
        // precalculate assignment factor
        shift = shift_c.x - (Scalar)i;
        shift_sq = shift*shift;
        shift_abs = copysignf(shift,Scalar(1.0));
        fac = (Scalar(3.0/2.0)-shift_abs);

        if (!i)
            {
            assign_x = Scalar(3.0/4.0)-shift_sq;
            assign_x_deriv = -Scalar(2.0)*shift;
            }
        else
            {
            assign_x = Scalar(1.0/2.0)*fac*fac;
            assign_x_deriv = -fac*shift/shift_abs;
            }

        for (int j = -1; j <= 1; ++j)
            {
            shift = shift_c.y - (Scalar)j;
            shift_sq = shift*shift;
            shift_abs = copysignf(shift,Scalar(1.0));
            fac = (Scalar(3.0/2.0)-shift_abs);

            if (!j)
                {
                assign_y = Scalar(3.0/4.0)-shift_sq;
                assign_y_deriv = -Scalar(2.0)*shift;
                }
            else
                {
                assign_y = Scalar(1.0/2.0)*fac*fac;
                assign_y_deriv = -fac*shift/shift_abs;
                }

            for (int k = -1; k <= 1; ++k)
                {
                shift = shift_c.z - (Scalar)k;
                shift_sq = shift*shift;
                shift_abs = copysignf(shift,Scalar(1.0));
                fac = (Scalar(3.0/2.0)-shift_abs);

                if (!k)
                    {
                    assign_z = Scalar(3.0/4.0)-shift_sq;
                    assign_z_deriv = -Scalar(2.0)*shift;
                    }
                else
                    {
                    assign_z = Scalar(1.0/2.0)*fac*fac;
                    assign_z_deriv = -fac*shift/shift_abs;
                    }

                int neighi = (int) cell_coord.x + i;
                int neighj = (int) cell_coord.y + j;
                int neighk = (int) cell_coord.z + k;

                if (! n_ghost_cells.x)
                    {
                    if (neighi == grid_dim.x)
                        neighi = 0;
                    else if (neighi < 0)
                        neighi += grid_dim.x;
                    }

                if (! n_ghost_cells.y)
                    {
                    if (neighj == grid_dim.y)
                        neighj = 0;
                    else if (neighj < 0)
                        neighj += grid_dim.y;
                    }

                if (! n_ghost_cells.z)
                    {
                    if (neighk == grid_dim.z)
                        neighk = 0;
                    else if (neighk < 0)
                        neighk += grid_dim.z;
                    }

                // compute fraction of particle density assigned to cell
                unsigned int cell_idx;

                // use row-major layout
                cell_idx = neighi + grid_dim.x * (neighj + grid_dim.y * neighk);

                hipfftComplex inv_mesh = tex1Dfetch(inv_fourier_mesh_tex,cell_idx);

                force += -(Scalar)inner_dim.x*b1*mode*assign_x_deriv*assign_y*assign_z*inv_mesh.x;
                force += -(Scalar)inner_dim.y*b2*mode*assign_x*assign_y_deriv*assign_z*inv_mesh.x;
                force += -(Scalar)inner_dim.z*b3*mode*assign_x*assign_y*assign_z_deriv*inv_mesh.x;
                }
            }
        } // end neighbor cells loop

    // Normalization
    force *= Scalar(2.0)/(Scalar)n_global;

    // Multiply with bias potential derivative
    force *= bias;

    d_force[idx] = make_scalar4(force.x,force.y,force.z,0.0);
    }

void gpu_compute_forces(const unsigned int N,
                        const Scalar4 *d_postype,
                        Scalar4 *d_force,
                        const Scalar bias,
                        const hipfftComplex *d_inv_fourier_mesh,
                        const uint3 grid_dim,
                        const uint3 n_ghost_cells,
                        const Scalar *d_mode,
                        const BoxDim& box,
                        const BoxDim& global_box,
                        const unsigned int n_global)
    {
    const unsigned int block_size = 512;

    // force mesh includes ghost cells
    unsigned int num_cells = grid_dim.x*grid_dim.y*grid_dim.z;
    inv_fourier_mesh_tex.normalized = false;
    inv_fourier_mesh_tex.filterMode = hipFilterModePoint;
    hipBindTexture(0, inv_fourier_mesh_tex, d_inv_fourier_mesh, sizeof(Scalar4)*num_cells);

    // compute local inverse lattice vectors
    Scalar3 a1 = box.getLatticeVector(0);
    Scalar3 a2 = box.getLatticeVector(1);
    Scalar3 a3 = box.getLatticeVector(2);

    Scalar V_box = box.getVolume();
    Scalar3 b1 = make_scalar3(a2.y*a3.z-a2.z*a3.y, a2.z*a3.x-a2.x*a3.z, a2.x*a3.y-a2.y*a3.x)/V_box;
    Scalar3 b2 = make_scalar3(a3.y*a1.z-a3.z*a1.y, a3.z*a1.x-a3.x*a1.z, a3.x*a1.y-a3.y*a1.x)/V_box;
    Scalar3 b3 = make_scalar3(a1.y*a2.z-a1.z*a2.y, a1.z*a2.x-a1.x*a2.z, a1.x*a2.y-a1.y*a2.x)/V_box;

    gpu_compute_forces_kernel<<<N/block_size+1,block_size>>>(N,
             d_postype,
             d_force,
             bias,
             grid_dim,
             n_ghost_cells,
             d_mode,
             box,
             global_box.getVolume(),
             n_global,
             b1,
             b2,
             b3);

    hipUnbindTexture(inv_fourier_mesh_tex);
    }

__global__ void kernel_calculate_cv_partial(
            int n_wave_vectors,
            Scalar *sum_partial,
            const hipfftComplex *d_fourier_mesh,
            const hipfftComplex *d_fourier_mesh_G,
            const bool exclude_dc)
    {
    extern __shared__ Scalar sdata[];

    unsigned int tidx = threadIdx.x;

    int j = blockIdx.x * blockDim.x + threadIdx.x;

    Scalar mySum = Scalar(0.0);

    if (j < n_wave_vectors) {
        if (! exclude_dc || j != 0)
            mySum = d_fourier_mesh[j].x * d_fourier_mesh_G[j].x + d_fourier_mesh[j].y * d_fourier_mesh_G[j].y;
        }

    sdata[tidx] = mySum;

   __syncthreads();

    // reduce the sum
    int offs = blockDim.x >> 1;
    while (offs > 0)
        {
        if (tidx < offs)
            {
            sdata[tidx] += sdata[tidx + offs];
            }
        offs >>= 1;
        __syncthreads();
        }

    // write result to global memeory
    if (tidx == 0)
       sum_partial[blockIdx.x] = sdata[0];
    }

__global__ void kernel_final_reduce_cv(Scalar* sum_partial,
                                       unsigned int nblocks,
                                       Scalar *sum)
    {
    extern __shared__ Scalar smem[];

    if (threadIdx.x == 0)
       *sum = Scalar(0.0);

    for (int start = 0; start< nblocks; start += blockDim.x)
        {
        __syncthreads();
        if (start + threadIdx.x < nblocks)
            smem[threadIdx.x] = sum_partial[start + threadIdx.x];
        else
            smem[threadIdx.x] = Scalar(0.0);

        __syncthreads();

        // reduce the sum
        int offs = blockDim.x >> 1;
        while (offs > 0)
            {
            if (threadIdx.x < offs)
                smem[threadIdx.x] += smem[threadIdx.x + offs];
            offs >>= 1;
            __syncthreads();
            }

         if (threadIdx.x == 0)
            {
            *sum += smem[0];
            }
        }
    }

void gpu_compute_cv(unsigned int n_wave_vectors,
                   Scalar *d_sum_partial,
                   Scalar *d_sum,
                   const hipfftComplex *d_fourier_mesh,
                   const hipfftComplex *d_fourier_mesh_G,
                   const unsigned int block_size,
                   const uint3 mesh_dim,
                   const bool exclude_dc)
    {
    unsigned int n_blocks = n_wave_vectors/block_size + 1;

    unsigned int shared_size = block_size * sizeof(Scalar);
    kernel_calculate_cv_partial<<<n_blocks, block_size, shared_size>>>(
               n_wave_vectors,
               d_sum_partial,
               d_fourier_mesh,
               d_fourier_mesh_G,
               exclude_dc);

    // calculate final sum of mesh values
    const unsigned int final_block_size = 512;
    shared_size = final_block_size*sizeof(Scalar);
    kernel_final_reduce_cv<<<1, final_block_size,shared_size>>>(d_sum_partial,
                                                                n_blocks,
                                                                d_sum);
    }

__global__ void kernel_calculate_virial_partial(
            int n_wave_vectors,
            Scalar *sum_virial_partial,
            const Scalar *d_mesh_virial)
    {
    extern __shared__ Scalar sdata[];

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tidx = threadIdx.x;

    Scalar mySum_xx = Scalar(0.0);
    Scalar mySum_xy = Scalar(0.0);
    Scalar mySum_xz = Scalar(0.0);
    Scalar mySum_yy = Scalar(0.0);
    Scalar mySum_yz = Scalar(0.0);
    Scalar mySum_zz = Scalar(0.0);

    if (j < n_wave_vectors)
        {
        mySum_xx = d_mesh_virial[0*n_wave_vectors+j];
        mySum_xy = d_mesh_virial[1*n_wave_vectors+j];
        mySum_xz = d_mesh_virial[2*n_wave_vectors+j];
        mySum_yy = d_mesh_virial[3*n_wave_vectors+j];
        mySum_yz = d_mesh_virial[4*n_wave_vectors+j];
        mySum_zz = d_mesh_virial[5*n_wave_vectors+j];
        }

    sdata[0*blockDim.x+tidx] = mySum_xx;
    sdata[1*blockDim.x+tidx] = mySum_xy;
    sdata[2*blockDim.x+tidx] = mySum_xz;
    sdata[3*blockDim.x+tidx] = mySum_yy;
    sdata[4*blockDim.x+tidx] = mySum_yz;
    sdata[5*blockDim.x+tidx] = mySum_zz;

   __syncthreads();

    // reduce the sum
    int offs = blockDim.x >> 1;
    while (offs > 0)
        {
        if (tidx < offs)
            {
            sdata[0*blockDim.x+tidx] += sdata[0*blockDim.x+tidx + offs];
            sdata[1*blockDim.x+tidx] += sdata[1*blockDim.x+tidx + offs];
            sdata[2*blockDim.x+tidx] += sdata[2*blockDim.x+tidx + offs];
            sdata[3*blockDim.x+tidx] += sdata[3*blockDim.x+tidx + offs];
            sdata[4*blockDim.x+tidx] += sdata[4*blockDim.x+tidx + offs];
            sdata[5*blockDim.x+tidx] += sdata[5*blockDim.x+tidx + offs];
            }
        offs >>= 1;
        __syncthreads();
        }

    // write result to global memory
    if (tidx == 0)
        {
        sum_virial_partial[0*gridDim.x+blockIdx.x] = sdata[0*blockDim.x];
        sum_virial_partial[1*gridDim.x+blockIdx.x] = sdata[1*blockDim.x];
        sum_virial_partial[2*gridDim.x+blockIdx.x] = sdata[2*blockDim.x];
        sum_virial_partial[3*gridDim.x+blockIdx.x] = sdata[3*blockDim.x];
        sum_virial_partial[4*gridDim.x+blockIdx.x] = sdata[4*blockDim.x];
        sum_virial_partial[5*gridDim.x+blockIdx.x] = sdata[5*blockDim.x];
        }
    }


__global__ void kernel_final_reduce_virial(Scalar* sum_virial_partial,
                                           unsigned int nblocks,
                                           Scalar *sum_virial)
    {
    extern __shared__ Scalar smem[];

    if (threadIdx.x == 0)
        {
        sum_virial[0] = Scalar(0.0);
        sum_virial[1] = Scalar(0.0);
        sum_virial[2] = Scalar(0.0);
        sum_virial[3] = Scalar(0.0);
        sum_virial[4] = Scalar(0.0);
        sum_virial[5] = Scalar(0.0);
        }

    for (int start = 0; start< nblocks; start += blockDim.x)
        {
        __syncthreads();
        if (start + threadIdx.x < nblocks)
            {
            smem[0*blockDim.x+threadIdx.x] = sum_virial_partial[0*nblocks+start+threadIdx.x];
            smem[1*blockDim.x+threadIdx.x] = sum_virial_partial[1*nblocks+start+threadIdx.x];
            smem[2*blockDim.x+threadIdx.x] = sum_virial_partial[2*nblocks+start+threadIdx.x];
            smem[3*blockDim.x+threadIdx.x] = sum_virial_partial[3*nblocks+start+threadIdx.x];
            smem[4*blockDim.x+threadIdx.x] = sum_virial_partial[4*nblocks+start+threadIdx.x];
            smem[5*blockDim.x+threadIdx.x] = sum_virial_partial[5*nblocks+start+threadIdx.x];
            }
        else
            {
            smem[0*blockDim.x+threadIdx.x] = Scalar(0.0);
            smem[1*blockDim.x+threadIdx.x] = Scalar(0.0);
            smem[2*blockDim.x+threadIdx.x] = Scalar(0.0);
            smem[3*blockDim.x+threadIdx.x] = Scalar(0.0);
            smem[4*blockDim.x+threadIdx.x] = Scalar(0.0);
            smem[5*blockDim.x+threadIdx.x] = Scalar(0.0);
            }

        __syncthreads();

        // reduce the sum
        int offs = blockDim.x >> 1;
        while (offs > 0)
            {
            if (threadIdx.x < offs)
                {
                smem[0*blockDim.x+threadIdx.x] += smem[0*blockDim.x+threadIdx.x + offs];
                smem[1*blockDim.x+threadIdx.x] += smem[1*blockDim.x+threadIdx.x + offs];
                smem[2*blockDim.x+threadIdx.x] += smem[2*blockDim.x+threadIdx.x + offs];
                smem[3*blockDim.x+threadIdx.x] += smem[3*blockDim.x+threadIdx.x + offs];
                smem[4*blockDim.x+threadIdx.x] += smem[4*blockDim.x+threadIdx.x + offs];
                smem[5*blockDim.x+threadIdx.x] += smem[5*blockDim.x+threadIdx.x + offs];
                }
            offs >>= 1;
            __syncthreads();
            }

         if (threadIdx.x == 0)
            {
            sum_virial[0] += smem[0*blockDim.x];
            sum_virial[1] += smem[1*blockDim.x];
            sum_virial[2] += smem[2*blockDim.x];
            sum_virial[3] += smem[3*blockDim.x];
            sum_virial[4] += smem[4*blockDim.x];
            sum_virial[5] += smem[5*blockDim.x];
            }
        }
    }

void gpu_compute_virial(unsigned int n_wave_vectors,
                   Scalar *d_sum_virial_partial,
                   Scalar *d_sum_virial,
                   const Scalar *d_mesh_virial,
                   const unsigned int block_size)
    {
    unsigned int n_blocks = n_wave_vectors/block_size + 1;

    unsigned int shared_size = 6* block_size * sizeof(Scalar);
    kernel_calculate_virial_partial<<<n_blocks, block_size, shared_size>>>(
               n_wave_vectors,
               d_sum_virial_partial,
               d_mesh_virial);

    // calculate final virial values 
    const unsigned int final_block_size = 512;
    shared_size = 6*final_block_size*sizeof(Scalar);
    kernel_final_reduce_virial<<<1, final_block_size,shared_size>>>(d_sum_virial_partial,
                                                                  n_blocks,
                                                                  d_sum_virial);
    }

__device__ Scalar convolution_kernel(Scalar ksq, Scalar qstarsq)
    {
//    return expf(-ksq/qstarsq*Scalar(1.0/2.0));
    Scalar knorm = sqrtf(ksq);
    Scalar k_cut = sqrtf(qstarsq);
    return Scalar(1.0)/(Scalar(1.0)+expf(Scalar(12.0)*(knorm/k_cut-Scalar(1.0))));
    }

template<bool local_fft>
__global__ void gpu_compute_influence_function_kernel(const uint3 mesh_dim,
                                          const unsigned int n_wave_vectors,
                                          const uint3 global_dim,
                                          Scalar *d_inf_f,
                                          Scalar3 *d_k,
                                          const Scalar3 b1,
                                          const Scalar3 b2,
                                          const Scalar3 b3,
                                          const Scalar qstarsq,
                                          const int3 *zero_modes,
                                          const unsigned int n_zero_modes,
                                          const uint3 pidx,
                                          const uint3 pdim)
    {
    unsigned int kidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (kidx >= n_wave_vectors) return;

    int l,m,n;
    if (local_fft)
        {
        // use row-major layout
        int ny = mesh_dim.y;
        int nx = mesh_dim.x;
        n = kidx/ny/nx;
        m = (kidx-n*ny*nx)/nx;
        l = kidx % nx;
        }
#ifdef ENABLE_MPI
    else
        {
        // local layout: row-major
        int ny = mesh_dim.y;
        int nx = mesh_dim.x;
        int n_local = kidx/ny/nx;
        int m_local = (kidx-n_local*ny*nx)/nx;
        int l_local = kidx % nx;

        // cyclic distribution
        l = l_local*pdim.x + pidx.x;
        m = m_local*pdim.y + pidx.y;
        n = n_local*pdim.z + pidx.z;
        }
#endif

    // compute Miller indices
    if (l >= (int)(global_dim.x/2 + global_dim.x%2))
        l -= (int) global_dim.x;
    if (m >= (int)(global_dim.y/2 + global_dim.y%2))
        m -= (int) global_dim.y;
    if (n >= (int)(global_dim.z/2 + global_dim.z%2))
        n -= (int) global_dim.z;

    bool zero = false;
    for (unsigned int i = 0; i < n_zero_modes; ++i)
        {
        if (zero_modes[i].x == l && zero_modes[i].y == m && zero_modes[i].z == n)
            {
            zero = true;
            break;
            }
        }

    Scalar val(0.0);
    Scalar3 kval = (Scalar)l*b1+(Scalar)m*b2+(Scalar)n*b3;

    if (!zero)
        {
        Scalar ksq = dot(kval,kval);
        val = convolution_kernel(ksq,qstarsq);
        }

    // write out result
    d_inf_f[kidx] = val;
    d_k[kidx] = kval;
    }

void gpu_compute_influence_function(const uint3 mesh_dim,
                                    const uint3 global_dim,
                                    Scalar *d_inf_f,
                                    Scalar3 *d_k,
                                    const BoxDim& global_box,
                                    const Scalar qstarsq,
                                    const int3 *d_zero_modes,
                                    const unsigned int n_zero_modes,
                                    const bool local_fft,
                                    const uint3 pidx,
                                    const uint3 pdim)
    {
    // compute reciprocal lattice vectors
    Scalar3 a1 = global_box.getLatticeVector(0);
    Scalar3 a2 = global_box.getLatticeVector(1);
    Scalar3 a3 = global_box.getLatticeVector(2);

    Scalar V_box = global_box.getVolume();
    Scalar3 b1 = Scalar(2.0*M_PI)*make_scalar3(a2.y*a3.z-a2.z*a3.y, a2.z*a3.x-a2.x*a3.z, a2.x*a3.y-a2.y*a3.x)/V_box;
    Scalar3 b2 = Scalar(2.0*M_PI)*make_scalar3(a3.y*a1.z-a3.z*a1.y, a3.z*a1.x-a3.x*a1.z, a3.x*a1.y-a3.y*a1.x)/V_box;
    Scalar3 b3 = Scalar(2.0*M_PI)*make_scalar3(a1.y*a2.z-a1.z*a2.y, a1.z*a2.x-a1.x*a2.z, a1.x*a2.y-a1.y*a2.x)/V_box;

    unsigned int num_wave_vectors = mesh_dim.x*mesh_dim.y*mesh_dim.z;

    unsigned int block_size = 512;
    unsigned int n_blocks = num_wave_vectors/block_size;
    if (num_wave_vectors % block_size) n_blocks += 1;

    if (local_fft)
        gpu_compute_influence_function_kernel<true><<<n_blocks, block_size>>>(mesh_dim,
                                                                              num_wave_vectors,
                                                                              global_dim,
                                                                              d_inf_f,
                                                                              d_k,
                                                                              b1,
                                                                              b2,
                                                                              b3,
                                                                              qstarsq,
                                                                              d_zero_modes,
                                                                              n_zero_modes,
                                                                              pidx,
                                                                              pdim);
    #ifdef ENABLE_MPI
    else
        gpu_compute_influence_function_kernel<false><<<n_blocks,block_size>>>(mesh_dim,
                                                                             num_wave_vectors,
                                                                             global_dim,
                                                                             d_inf_f,
                                                                             d_k,
                                                                             b1,
                                                                             b2,
                                                                             b3,
                                                                             qstarsq,
                                                                             d_zero_modes,
                                                                             n_zero_modes,
                                                                             pidx,
                                                                             pdim);
    #endif
    }

__global__ void gpu_compute_qmax_partial_kernel(
            int n_wave_vectors,
            Scalar4 *max_partial,
            const Scalar3 *d_k,
            const hipfftComplex *d_fourier_mesh)
    {
    extern __shared__ Scalar4 sdata_max[];

    unsigned int tidx = threadIdx.x;

    int j = blockIdx.x * blockDim.x + threadIdx.x;

    Scalar4 max_q = make_scalar4(0.0,0.0,0.0,0.0);

    if (j < n_wave_vectors) {
        Scalar a = d_fourier_mesh[j].x * d_fourier_mesh[j].x + d_fourier_mesh[j].y * d_fourier_mesh[j].y;
        Scalar3 k = d_k[j];
        max_q = make_scalar4(k.x,k.y,k.z,a);
        }

    sdata_max[tidx] = max_q;

   __syncthreads();

    // reduce the sum
    int offs = blockDim.x >> 1;
    while (offs > 0)
        {
        if (tidx < offs)
            {
            sdata_max[tidx] = (sdata_max[tidx].w > sdata_max[tidx + offs].w) ? sdata_max[tidx] : sdata_max[tidx + offs];
            }
        offs >>= 1;
        __syncthreads();
        }

    // write result to global memeory
    if (tidx == 0)
       max_partial[blockIdx.x] = sdata_max[0];
    }

__global__ void gpu_compute_qmax_final_kernel(Scalar4* max_partial,
                                       unsigned int nblocks,
                                       Scalar4 *q_max)
    {
    extern __shared__ Scalar4 sdata_max[];

    if (threadIdx.x == 0)
       *q_max = make_scalar4(0.0,0.0,0.0,0.0);

    for (int start = 0; start< nblocks; start += blockDim.x)
        {
        __syncthreads();
        if (start + threadIdx.x < nblocks)
            sdata_max[threadIdx.x] = max_partial[start + threadIdx.x];
        else
            sdata_max[threadIdx.x] = make_scalar4(0.0,0.0,0.0,0.0);

        __syncthreads();

        // reduce the sum
        int offs = blockDim.x >> 1;
        while (offs > 0)
            {
            if (threadIdx.x < offs)
                sdata_max[threadIdx.x] = (sdata_max[threadIdx.x].w > sdata_max[threadIdx.x + offs].w) ?
                                         sdata_max[threadIdx.x] : sdata_max[threadIdx.x + offs];
            offs >>= 1;
            __syncthreads();
            }

         if (threadIdx.x == 0)
            {
            Scalar4 old_qmax = *q_max;
            *q_max = (old_qmax.w > sdata_max[0].w) ? old_qmax : sdata_max[0];
            }
        }
    }

void gpu_compute_q_max(unsigned int n_wave_vectors,
                   Scalar4 *d_max_partial,
                   Scalar4 *d_q_max,
                   const Scalar3 *d_k,
                   const hipfftComplex *d_fourier_mesh,
                   const unsigned int block_size)
    {
    unsigned int n_blocks = n_wave_vectors/block_size + 1;

    unsigned int shared_size = block_size * sizeof(Scalar4);
    gpu_compute_qmax_partial_kernel<<<n_blocks, block_size, shared_size>>>(
               n_wave_vectors,
               d_max_partial,
               d_k,
               d_fourier_mesh);

    // calculate final sum of mesh values
    const unsigned int final_block_size = 512;
    shared_size = final_block_size*sizeof(Scalar4);
    gpu_compute_qmax_final_kernel<<<1, final_block_size,shared_size>>>(d_max_partial,
                                                                n_blocks,
                                                                d_q_max);
    }
